#include <hip/hip_runtime.h>

// CUDA kernel for ReLU activation function
__global__ void reluKernel(float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = fmaxf(0.0, input[idx]);
    }
}

// Wrapper function to call the CUDA kernel
void reluLauncher(float* input, float* output, int size) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    reluKernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, size);

    hipDeviceSynchronize();
}

__global__ void layerNormKernel(float* input, float* output, int B, int N, int D, float* gamma, float* beta) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int BN = B * N;
    if (idx < BN) {
        float* offset_in = input + idx * D;

        // calculate mean
        float mean = 0;
        for (int i = 0; i < D; i++) {
            mean += *(offset_in + i);
        }
        mean /= D;

        // calculate std
        float var = 0;
        for (int i = 0; i < D; i++) {
            var += pow(*(offset_in + i) - mean, 2);
        }
        var /= D;
        float std_inv = 1.0 / sqrt(var + 1e-5);

        // normalize
        float* offset_out = output + idx * D; 
        for (int i = 0; i < D; i++) {
            float out_i = (offset_in[i] - mean) * std_inv;
            offset_out[i] = out_i * gamma[i] + beta[i];
        }
    }
}

void layerNormLauncher(float* input, float* output, int B, int N, int D, float* gamma, float* beta) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (B * N + threadsPerBlock - 1) / threadsPerBlock;

    layerNormKernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, B, N, D, gamma, beta);

}


__global__ void softmaxKernel(float* input, float* output, int B, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int BN = B * N;
    if (idx < BN) {
        float* offset_in = input + idx * N;
        float sum = 0.0;
        float max = -INFINITY;

        for (int i = 0; i < N; i++) {
            max = fmaxf(max, offset_in[i]);
        }

        for (int i = 0; i < N; i++) {
            sum += exp(offset_in[i] - max);
        }

        float* offset_out = output + idx * N;
        for (int i = 0; i < N; i++) {
            offset_out[i] = exp(offset_in[i] - max) / sum;
        }
    }
}

void softmaxLauncher(float* input, float* output, int B, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (B * N + threadsPerBlock - 1) / threadsPerBlock;

    softmaxKernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, B, N);
}

// TODO: Layer norm
// TODO: RMS norm
// TODO: Attention (Q, K, V proj, softmax, matmul, outproj)
// TODO: Matmul linear 1
// TODO: activations -- Swiglu, Relu, GeLU
// TODO: Matmul linear 2
// TODO: residual
